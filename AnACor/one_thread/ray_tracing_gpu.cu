#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iomanip>
#include <cassert>
#include <cmath>
#include <stdio.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"
#include "GPU_reduction.cuh"

#define DEBUG 0

#include "timer.h"

#define warpSize 32
#define INDEX_3D(N3, N2, N1, I3, I2, I1) (N1 * (N2 * I3 + I2) + I1)

__device__ __constant__ int x_max, y_max, z_max, diagonal, len_coord_list, len_result;
__device__ __constant__ float coeff_cr, coeff_bu, coeff_lo, coeff_li, voxel_length_x, voxel_length_y, voxel_length_z;

void print_cuda_error(hipError_t code)
{
	printf("CUDA error code: %d; string: %s;\n", (int)code, hipGetErrorString(code));
}

__global__ void rt_gpu_python_results(float *d_result_list, float *d_python_result_list, int h_len_result)
{
	size_t id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < h_len_result)
	{
		float gpu_absorption = 0;
		for (int j = 0; j < len_coord_list; j++)
		{
			gpu_absorption += exp(-(d_result_list[id * len_coord_list * 2 + 2 * j + 0] + d_result_list[id * len_coord_list * 2 + 2 * j + 1]));
		}
		float gpu_absorption_mean = gpu_absorption / ((float)len_coord_list);
		d_python_result_list[id] = gpu_absorption_mean;
	}
}

__inline__ __device__ void transpose_device(float *input, int rows, int cols, float *output)
{
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			output[j * rows + i] = input[i * cols + j];
		}
	}
}

__inline__ __device__ void dot_product_device(const float *A, const float *B, float *C, int m, int n, int p)
{
	//     In the provided example, the dimensions m, n, and p of the matrices are as follows:

	// Matrix A: m x n = 2 x 3 (2 rows, 3 columns)
	// Matrix B: n x p = 3 x 2 (3 rows, 2 columns)
	// Matrix C: m x p = 2 x 2 (2 rows, 2 columns)
	for (int i = 0; i < m; i++)
	{
		for (int j = 0; j < p; j++)
		{
			float sum = 0.0;
			for (int k = 0; k < n; k++)
			{
				sum += A[i * n + k] * B[k * p + j];
			}
			C[i * p + j] = sum;
		}
	}
}

__inline__ __device__ void kp_rotation_device(const float *axis, float theta, float *result)
{
	float x = axis[0];
	float y = axis[1];
	float z = axis[2];
	float c = cosf(theta);
	float s = sinf(theta);

	result[0] = c + (x * x) * (1 - c);
	result[1] = x * y * (1 - c) - z * s;
	result[2] = y * s + x * z * (1 - c);

	result[3] = z * s + x * y * (1 - c);
	result[4] = c + (y * y) * (1 - c);
	result[5] = -x * s + y * z * (1 - c);

	result[6] = -y * s + x * z * (1 - c);
	result[7] = x * s + y * z * (1 - c);
	result[8] = c + (z * z) * (1 - c);
}

__global__ void ray_tracing_rotation(const float *d_omega_axis, float *d_omega_list, float *d_kp_rotation_matrix, float *d_raw_xray, float *d_scattering_vector_list,  float *d_rotated_xray_list, float *d_rotated_s1_list)
{
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;
	float rotation_matrix_frame_omega[9];
	float rotation_matrix_overall[9];
	float total_rotation_matrix[9];
	float rotated_xray[3];
	float rotated_s1[3];
	if (id < len_result)
	{
		kp_rotation_device(d_omega_axis, d_omega_list[id], rotation_matrix_frame_omega);
		dot_product_device((float *)rotation_matrix_frame_omega, d_kp_rotation_matrix, (float *)rotation_matrix_overall, 3, 3, 3);
		transpose_device((float *)rotation_matrix_overall, 3, 3, (float *)total_rotation_matrix);

		dot_product_device((float *)total_rotation_matrix, d_raw_xray, (float *)rotated_xray, 3, 3, 1);
		d_rotated_xray_list[3 * id] = rotated_xray[0];
		d_rotated_xray_list[3 * id + 1] = rotated_xray[1];
		d_rotated_xray_list[3 * id + 2] = rotated_xray[2];

		float scattering_vector[3] = {d_scattering_vector_list[id * 3],
									  d_scattering_vector_list[id * 3 + 1],
									  d_scattering_vector_list[id * 3 + 2]};
		dot_product_device((float *)total_rotation_matrix, (float *)scattering_vector, (float *)rotated_s1, 3, 3, 1);
		d_rotated_s1_list[3 * id] = rotated_s1[0];
		d_rotated_s1_list[3 * id + 1] = rotated_s1[1];
		d_rotated_s1_list[3 * id + 2] = rotated_s1[2];
	}
}

__inline__ __device__ int cube_face(int *ray_origin, float *ray_direction, int L1)
{
	float t_min = x_max * y_max * z_max, dtemp = 0;
	int face_id = 0;

	// float tx_min = (min_x - ray_origin[2]) / ray_direction[2];
	//  dtemp = (0 - ray_origin[2]) / ray_direction[2];
	if (L1)
	{
		dtemp = -(0 - ray_origin[2]) / ray_direction[2];
	}
	else
	{
		dtemp = (0 - ray_origin[2]) / ray_direction[2];
	}
	if (dtemp >= 0)
	{
		t_min = dtemp;
		face_id = 1;
	}

	// float tx_max = (max_x - ray_origin[2]) / ray_direction[2];
	if (L1)
	{
		dtemp = -(x_max - ray_origin[2]) / ray_direction[2];
	}
	else
	{
		dtemp = (x_max - ray_origin[2]) / ray_direction[2];
	}
	// dtemp = (x_max - ray_origin[2]) / ray_direction[2];
	if (dtemp >= 0 && dtemp < t_min)
	{
		t_min = dtemp;
		face_id = 2;
	}

	// float ty_min = (min_y - ray_origin[1]) / ray_direction[1];
	//  dtemp = (0 - ray_origin[1]) / ray_direction[1];
	if (L1)
	{
		dtemp = -(0 - ray_origin[1]) / ray_direction[1];
	}
	else
	{
		dtemp = (0 - ray_origin[1]) / ray_direction[1];
	}
	if (dtemp >= 0 && dtemp < t_min)
	{
		t_min = dtemp;
		face_id = 3;
	}

	// float ty_max = (max_y - ray_origin[1]) / ray_direction[1];
	//  dtemp = (y_max - ray_origin[1]) / ray_direction[1];
	if (L1)
	{
		dtemp = -(y_max - ray_origin[1]) / ray_direction[1];
	}
	else
	{
		dtemp = (y_max - ray_origin[1]) / ray_direction[1];
	}
	if (dtemp >= 0 && dtemp < t_min)
	{
		t_min = dtemp;
		face_id = 4;
	}
	// float tz_min = (min_z - ray_origin[0]) / ray_direction[0];
	//  dtemp = (0 - ray_origin[0]) / ray_direction[0];
	if (L1)
	{
		dtemp = -(0 - ray_origin[0]) / ray_direction[0];
	}
	else
	{
		dtemp = (0 - ray_origin[0]) / ray_direction[0];
	}
	if (dtemp >= 0 && dtemp < t_min)
	{
		t_min = dtemp;
		face_id = 5;
	}

	// float tz_max = (max_z - ray_origin[0]) / ray_direction[0];
	//  dtemp = (z_max - ray_origin[0]) / ray_direction[0];
	if (L1)
	{
		dtemp = -(z_max - ray_origin[0]) / ray_direction[0];
	}
	else
	{
		dtemp = (z_max - ray_origin[0]) / ray_direction[0];
	}
	if (dtemp >= 0 && dtemp < t_min)
	{
		t_min = dtemp;
		face_id = 6;
	}

	if (face_id == 1)
	{ // tx_min
		return 6;
	}
	else if (face_id == 2)
	{ // tx_max
		return 1;
	}
	else if (face_id == 3)
	{ // 3 ty_min
		return 4;
	}
	else if (face_id == 4)
	{ // 4 ty_max
		return 5;
	}
	else if (face_id == 5)
	{ // 5 tz_min
		return 2;
	}
	else if (face_id == 6)
	{ // 6 tz_max
		return 3;
	}
	else
	{
		return 0;
	}
}

__global__ void rt_gpu_get_face_overall(int *d_face, int *d_coord_list, float *d_rotated_s1_list, float *d_rotated_xray_list)
{
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;
	size_t batch_number = blockIdx.y * blockDim.y + threadIdx.y;
	int is_ray_incomming = id & 1;
	size_t pos = (id >> 1);
	// if (threadIdx.x==3){
	// printf("batch_number=%d, id=%ld, blockIdx.x=%d ,blockDim.x=%d, threadIdx.x=%d, blockIdx.y=%d, blockDim.y=%d, threadIdx.y=%d, is_ray_incomming=%d, pos=%d\n", batch_number, id, blockIdx.x, blockDim.x, threadIdx.x, blockIdx.y, blockDim.y, threadIdx.y, is_ray_incomming, pos);
	// }
	int coord[3];
	float ray_direction[3];
	// printf("batch number=%d, len_result=%d\n", batch_number, len_result);
	if (batch_number < len_result)
	{
		if (pos < len_coord_list)
		{
			coord[0] = d_coord_list[3 * pos + 0];
			coord[1] = d_coord_list[3 * pos + 1];
			coord[2] = d_coord_list[3 * pos + 2];

			if (is_ray_incomming == 1)
			{
				ray_direction[0] = d_rotated_xray_list[batch_number * 3 + 0];
				ray_direction[1] = d_rotated_xray_list[batch_number * 3 + 2];
				ray_direction[2] = d_rotated_xray_list[batch_number * 3 + 1];
			}
			else
			{
				ray_direction[0] = d_rotated_s1_list[batch_number * 3 + 0];
				ray_direction[1] = d_rotated_s1_list[batch_number * 3 + 2];
				ray_direction[2] = d_rotated_s1_list[batch_number * 3 + 1];
			}
			int face = cube_face(coord, ray_direction, is_ray_incomming);
			// printf("face=%d\n", face);

			d_face[batch_number * len_coord_list * 2 + id] = face;
		}
	}
}


__global__ void rt_gpu_get_face(int *d_face, int *d_coord_list, float *d_rotated_s1_list, float *d_rotated_xray_list,  int batch_number)
{
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;
	int is_ray_incomming = id & 1;
	size_t pos = (id >> 1);

	int coord[3];
	float ray_direction[3];

	if (pos < len_coord_list)
	{
		coord[0] = d_coord_list[3 * pos + 0];
		coord[1] = d_coord_list[3 * pos + 1];
		coord[2] = d_coord_list[3 * pos + 2];

		if (is_ray_incomming == 1)
		{
			ray_direction[0] = d_rotated_xray_list[batch_number * 3 + 0];
			ray_direction[1] = d_rotated_xray_list[batch_number * 3 + 2];
			ray_direction[2] = d_rotated_xray_list[batch_number * 3 + 1];
		}
		else
		{
			ray_direction[0] = d_rotated_s1_list[batch_number * 3 + 0];
			ray_direction[1] = d_rotated_s1_list[batch_number * 3 + 2];
			ray_direction[2] = d_rotated_s1_list[batch_number * 3 + 1];
		}
		int face = cube_face(coord, ray_direction,is_ray_incomming);
		d_face[id] = face;
	}
}



__inline__ __device__ void get_theta_phi(float *theta, float *phi, float *ray_direction, int L1)
{
	if (L1 == 1)
	{
		ray_direction[0] = -ray_direction[0];
		ray_direction[1] = -ray_direction[1];
		ray_direction[2] = -ray_direction[2];
	}

	if (ray_direction[1] == 0)
	{
		(*theta) = atanf(-ray_direction[2] / (-sqrtf(ray_direction[0] * ray_direction[0] + ray_direction[1] * ray_direction[1]) + 0.001));
		(*phi) = atanf(-ray_direction[0] / (ray_direction[1] + 0.001));
	}
	else
	{
		if (ray_direction[1] < 0)
		{
			(*theta) = atanf(-ray_direction[2] / sqrtf(ray_direction[0] * ray_direction[0] + ray_direction[1] * ray_direction[1]));
			(*phi) = atanf(-ray_direction[0] / (ray_direction[1]));
		}
		else
		{
			if (ray_direction[2] < 0)
			{
				(*theta) = M_PI - atanf(-ray_direction[2] / sqrtf(ray_direction[0] * ray_direction[0] + ray_direction[1] * ray_direction[1]));
			}
			else
			{
				(*theta) = -M_PI - atanf(-ray_direction[2] / sqrtf(ray_direction[0] * ray_direction[0] + ray_direction[1] * ray_direction[1]));
			}
			(*phi) = -atanf(-ray_direction[0] / (-ray_direction[1]));
		}
	}
}

__global__ void rt_gpu_angles(float *d_angles, float *d_rotated_s1_list, float *d_rotated_xray_list, int nBatches, int batch_number)
{
	size_t id = blockIdx.x * blockDim.x + threadIdx.x;
	size_t batch = (id >> 1);
	int is_ray_incomming = id & 1;

	float theta = 0, phi = 0;
	float ray_direction[3];

	if (batch < nBatches)
	{
		if (is_ray_incomming == 1)
		{
			ray_direction[0] = d_rotated_xray_list[batch_number * 3 + 0];
			ray_direction[1] = d_rotated_xray_list[batch_number * 3 + 1];
			ray_direction[2] = d_rotated_xray_list[batch_number * 3 + 2];
		}
		else
		{
			ray_direction[0] = d_rotated_s1_list[batch_number * 3 + 0];
			ray_direction[1] = d_rotated_s1_list[batch_number * 3 + 1];
			ray_direction[2] = d_rotated_s1_list[batch_number * 3 + 2];
		}

		get_theta_phi(&theta, &phi, ray_direction, is_ray_incomming);

		// printf("pos=[%d; %d] theta=%f; phi=%f;\n", (int) (2*id + 0), (int) (2*id + 1), theta, phi);

		d_angles[2 * id + 0] = theta;
		d_angles[2 * id + 1] = phi;
	}
	// printf("d_angles =[%f,%f,%f,%f]  ", d_angles[0], d_angles[1], d_angles[2], d_angles[3]);
}

__inline__ __device__ void get_increment_ratio(
	float *increment_ratio_x,
	float *increment_ratio_y,
	float *increment_ratio_z,
	float theta,
	float phi,
	int face)
{
	if (face == 1)
	{
		*increment_ratio_x = -1;
		*increment_ratio_y = tanf(M_PI - theta) / cosf(fabs(phi));
		*increment_ratio_z = tanf(phi);
	}
	else if (face == 2)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*increment_ratio_x = 1 / tanf(fabs(phi));
			*increment_ratio_y = tanf(theta) / sinf(fabs(phi));
			*increment_ratio_z = -1;
		}
		else
		{
			*increment_ratio_x = 1 / tanf(fabs(phi));
			*increment_ratio_y = tanf(M_PI - theta) / sinf(fabs(phi));
			*increment_ratio_z = -1;
		}
	}
	else if (face == 3)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*increment_ratio_x = 1 / tanf(fabs(phi));
			*increment_ratio_y = tanf(theta) / sinf(fabs(phi));
			*increment_ratio_z = 1;
		}
		else
		{
			*increment_ratio_x = 1 / (tanf(fabs(phi)));
			*increment_ratio_y = tanf(M_PI - theta) / sinf(fabs(phi));
			*increment_ratio_z = 1;
		}
	}
	else if (face == 4)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*increment_ratio_x = cosf(fabs(phi)) / tanf(fabs(theta));
			*increment_ratio_y = 1;
			*increment_ratio_z = sinf(phi) / tanf(fabs(theta));
		}
		else
		{
			*increment_ratio_x = cosf(fabs(phi)) / (tanf((M_PI - fabs(theta))));
			*increment_ratio_y = 1;
			*increment_ratio_z = sinf(-phi) / (tanf((M_PI - fabs(theta))));
		}
	}
	else if (face == 5)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*increment_ratio_x = cosf(fabs(phi)) / (tanf(fabs(theta)));
			*increment_ratio_y = -1;
			*increment_ratio_z = sinf(phi) / (tanf(fabs(theta)));
		}
		else
		{
			*increment_ratio_x = cosf(fabs(phi)) / (tanf(M_PI - fabs(theta)));
			*increment_ratio_y = -1;
			*increment_ratio_z = sinf(phi) / (tanf(M_PI - fabs(theta)));
		}
	}
	else if (face == 6)
	{
		*increment_ratio_x = -1;
		*increment_ratio_y = tanf(theta) / cosf(phi);
		*increment_ratio_z = tanf(phi);
	}
}

__global__ void rt_gpu_increments(float *d_increments, float *d_angles)
{
	// store increments according to different faces and different thetas
	// and for one single reflection, the increments are the same
	// so we only need to store the increments for one single reflection with
	// different crystal voxel positions
	size_t id = threadIdx.x;
	size_t batch = blockIdx.x;
	int face = id % 6;
	int is_ray_incomming = id / 6.0f;

	float theta = 0, phi = 0;
	if (is_ray_incomming == 1)
	{
		theta = d_angles[4 * batch + 2 + 0];
		phi = d_angles[4 * batch + 2 + 1];
	}
	else
	{
		theta = d_angles[4 * batch + 0];
		phi = d_angles[4 * batch + 1];
	}

	float ix = 0, iy = 0, iz = 0;
	get_increment_ratio(&ix, &iy, &iz, theta, phi, face + 1);

	d_increments[36 * batch + 3 * threadIdx.x + 0] = ix;
	d_increments[36 * batch + 3 * threadIdx.x + 1] = iy;
	d_increments[36 * batch + 3 * threadIdx.x + 2] = iz;
}

__inline__ __device__ void get_new_coordinates(
	int *new_x, int *new_y, int *new_z,
	int x, int y, int z,
	float increment_ratio_x, float increment_ratio_y, float increment_ratio_z,
	int increment, float theta, int face)
{
	if (face == 1)
	{
		if (theta > 0)
		{
			// this -1 represents that the opposition of direction
			// between the lab x-axis and the wavevector
			*new_x = (int)(x - increment * increment_ratio_x);
			*new_y = (int)(y - increment * increment_ratio_y);
			*new_z = (int)(z - increment * increment_ratio_z);
		}
		else
		{
			// this -1 represents that the opposition of direction
			// between the lab x-axis and the wavevector
			*new_x = (int)(x - increment * increment_ratio_x + 0.5);
			*new_y = (int)(y - increment * increment_ratio_y + 0.5);
			*new_z = (int)(z - increment * increment_ratio_z + 0.5);
		}
	}
	else if (face == 2)
	{
		if (fabs(theta) < M_PI / 2)
		{
			if (theta > 0)
			{
				*new_x = (int)(x + -1 * increment * increment_ratio_x);
				*new_y = (int)(y - increment * increment_ratio_y);
				*new_z = (int)(z + increment * increment_ratio_z);
			}
			else
			{
				*new_x = (int)(x + -1 * increment * increment_ratio_x + 0.5);
				*new_y = (int)(y - increment * increment_ratio_y + 0.5);
				*new_z = (int)(z + increment * increment_ratio_z + 0.5);
			}
		}
		else
		{
			if (theta > 0)
			{
				*new_x = (int)(x + 1 * increment * increment_ratio_x);
				*new_y = (int)(y - increment * increment_ratio_y);
				*new_z = (int)(z + increment * increment_ratio_z);
			}
			else
			{
				*new_x = (int)(x + 1 * increment * increment_ratio_x + 0.5);
				*new_y = (int)(y - increment * increment_ratio_y + 0.5);
				*new_z = (int)(z + increment * increment_ratio_z + 0.5);
			}
		}
	}
	else if (face == 3)
	{
		if (fabs(theta) < M_PI / 2)
		{
			if (theta > 0)
			{
				*new_x = (int)(x + -1 * increment * increment_ratio_x);
				*new_y = (int)(y - increment * increment_ratio_y);
				*new_z = (int)(z + increment * increment_ratio_z);
			}
			else
			{
				*new_x = (int)(x + -1 * increment * increment_ratio_x + 0.5);
				*new_y = (int)(y - increment * increment_ratio_y + 0.5);
				*new_z = (int)(z + increment * increment_ratio_z + 0.5);
			}
		}
		else
		{
			if (theta > 0)
			{
				*new_x = (int)(x + 1 * increment * increment_ratio_x);
				*new_y = (int)(y - increment * increment_ratio_y);
				*new_z = (int)(z + increment * 1);
			}
			else
			{
				*new_x = (int)(x + 1 * increment * increment_ratio_x + 0.5);
				*new_y = (int)(y - increment * increment_ratio_y + 0.5);
				*new_z = (int)(z + increment * 1 + 0.5);
			}
		}
	}
	else if (face == 4)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*new_x = (int)(x + -1 * increment * increment_ratio_x);
			*new_y = (int)(y - increment * increment_ratio_y);
			*new_z = (int)(z + increment * increment_ratio_z);
		}
		else
		{
			*new_x = (int)(x + 1 * increment * increment_ratio_x);
			*new_y = (int)(y - increment * increment_ratio_y);
			*new_z = (int)(z + increment * increment_ratio_z);
		}
	}
	else if (face == 5)
	{
		if (fabs(theta) < M_PI / 2)
		{
			*new_x = (int)(x + -1 * increment * increment_ratio_x + 0.5);
			*new_y = (int)(y - increment * increment_ratio_y + 0.5);
			*new_z = (int)(z + increment * increment_ratio_z + 0.5);
		}
		else
		{
			*new_x = (int)(x + 1 * increment * increment_ratio_x + 0.5);
			*new_y = (int)(y - increment * increment_ratio_y + 0.5);
			*new_z = (int)(z - increment * increment_ratio_z + 0.5);
		}
	}
	else if (face == 6)
	{
		if (theta > 0)
		{
			*new_x = (int)(x + increment * increment_ratio_x);
			*new_y = (int)(y - increment * increment_ratio_y);
			*new_z = (int)(z + increment * increment_ratio_z);
		}
		else
		{
			*new_x = (int)(x + increment * increment_ratio_x + 0.5);
			*new_y = (int)(y - increment * increment_ratio_y + 0.5);
			*new_z = (int)(z + increment * increment_ratio_z + 0.5);
		}
	}
}

__inline__ __device__ void get_distance_2(float *total_length, float s_sum, float increment_ratio_x, float increment_ratio_y, float increment_ratio_z, int face)
{
	float dist_x, dist_y, dist_z;
	if (face == 1)
	{
		dist_x = (s_sum - 1.0f);
		dist_y = (s_sum - 1.0f) * increment_ratio_y;
		dist_z = (s_sum - 1.0f) * increment_ratio_z;
	}
	else if (face == 2)
	{
		dist_x = (s_sum - 1.0f) * increment_ratio_x;
		dist_y = (s_sum - 1.0f) * increment_ratio_y;
		dist_z = (s_sum - 1.0f);
	}
	else if (face == 3)
	{
		dist_x = (s_sum - 1.0f) * increment_ratio_x;
		dist_y = (s_sum - 1.0f) * increment_ratio_y;
		dist_z = (s_sum - 1.0f);
	}
	else if (face == 4)
	{
		dist_x = (s_sum - 1.0f) * increment_ratio_x;
		dist_y = (s_sum - 1.0f);
		dist_z = (s_sum - 1.0f) * increment_ratio_z;
	}
	else if (face == 5)
	{
		dist_x = (s_sum - 1.0f) * increment_ratio_x;
		dist_y = (s_sum - 1.0f);
		dist_z = (s_sum - 1.0f) * increment_ratio_z;
	}
	else if (face == 6)
	{
		dist_x = (s_sum - 1.0f);
		dist_y = (s_sum - 1.0f) * increment_ratio_y;
		dist_z = (s_sum - 1.0f) * increment_ratio_z;
	}
	// 	if (face == 1)
	// {
	// 	dist_x = (s_sum  );
	// 	dist_y = (s_sum  ) * increment_ratio_y;
	// 	dist_z = (s_sum  ) * increment_ratio_z;
	// }
	// else if (face == 2)
	// {
	// 	dist_x = (s_sum  ) * increment_ratio_x;
	// 	dist_y = (s_sum  ) * increment_ratio_y;
	// 	dist_z = (s_sum  );
	// }
	// else if (face == 3)
	// {
	// 	dist_x = (s_sum  ) * increment_ratio_x;
	// 	dist_y = (s_sum  ) * increment_ratio_y;
	// 	dist_z = (s_sum  );
	// }
	// else if (face == 4)
	// {
	// 	dist_x = (s_sum  ) * increment_ratio_x;
	// 	dist_y = (s_sum  );
	// 	dist_z = (s_sum  ) * increment_ratio_z;
	// }
	// else if (face == 5)
	// {
	// 	dist_x = (s_sum  ) * increment_ratio_x;
	// 	dist_y = (s_sum  );
	// 	dist_z = (s_sum  ) * increment_ratio_z;
	// }
	// else if (face == 6)
	// {
	// 	dist_x = (s_sum  );
	// 	dist_y = (s_sum  ) * increment_ratio_y;
	// 	dist_z = (s_sum  ) * increment_ratio_z;
	// }
	else
	{
		dist_x = 0;
		dist_y = 0;
		dist_z = 0;
	}
	// if (id <2){
	// printf("id: %d dist_x: %f, dist_y: %f, dist_z: %f\n",id, dist_x, dist_y, dist_z);
	// }
	*total_length = sqrtf(
		(dist_x * voxel_length_x) * (dist_x * voxel_length_x) +
		(dist_y * voxel_length_y) * (dist_y * voxel_length_y) +
		(dist_z * voxel_length_z) * (dist_z * voxel_length_z));
}

__global__ void rt_gpu_absorption(int8_t *d_label_list, int *d_coord_list, int *d_face, float *d_angles, float *d_increments, float *d_result_list, int index)
{
	size_t id = blockIdx.x;
	int is_ray_incomming = id & 1;
	size_t pos = (id >> 1); /* the right shift operation effectively divided the value of id by 2 (since shifting the bits to the right by 1 is equivalent to integer division by 2).*/
	float increments[3];
	int face = 0;
	int coord[3];
	float theta, phi;
	__shared__ float s_absorption[1024];
	// __shared__ int s_ray_classes[512];
	int cr_l_2_int = 0;
	int li_l_2_int = 0;
	int bu_l_2_int = 0;
	int lo_l_2_int = 0;

	// Load coordinates
	coord[0] = d_coord_list[3 * pos + 0]; // z
	coord[1] = d_coord_list[3 * pos + 1]; // y
	coord[2] = d_coord_list[3 * pos + 2]; // x

	// Load face
	//face = d_face[index * len_coord_list * 2 + id];
	face = d_face[id];
	// printf("index= %ld face=%d\n", index*len_coord_list*2 + id,face);

	// Load angle
	theta = d_angles[4 * blockIdx.y + 2 * is_ray_incomming];
	// phi = d_angles[4*blockIdx.y + 2*is_ray_incomming + 1];

	// Load Increment
	size_t incr_pos = 36 * blockIdx.y + 18 * is_ray_incomming + 3 * (face - 1);
	// get_increment_ratio(&increments[0], &increments[1], &increments[2], theta, phi, face);
	increments[0] = d_increments[incr_pos + 0];
	increments[1] = d_increments[incr_pos + 1];
	increments[2] = d_increments[incr_pos + 2];

	// Calculate number of iterations of blocks
	// trick for ceiling
	int nIter = (int)((diagonal + blockDim.x - 1) / blockDim.x);

	for (int f = 0; f < nIter; f++)
	{
		// calculate position based on threads id
		// check if the position is within a cube_face
		// write into ray_direction
		int lpos = (f * blockDim.x + threadIdx.x);
		int x, y, z;
		get_new_coordinates(
			&x, &y, &z,
			coord[2], coord[1], coord[0],
			increments[0], increments[1], increments[2],
			lpos, theta, face);
		int label = 0;

		if (
			x < x_max && y < y_max && z < z_max &&
			x >= 0 && y >= 0 && z >= 0)
		{
			size_t cube_pos = INDEX_3D(
				z_max, y_max, x_max,
				z, y, x);
			label = (int)d_label_list[cube_pos];

			if (label == 3)
				cr_l_2_int++;
			else if (label == 1)
				li_l_2_int++;
			else if (label == 2)
				lo_l_2_int++;
			else if (label == 4)
				bu_l_2_int++;
			else
			{
			}
		}
		// if (lpos < diagonal)
		// {
		// 	size_t gpos = blockIdx.x * diagonal + lpos;
		// 	d_ray_classes[gpos] = label;
		// }
	}

	float total_length;
	get_distance_2(&total_length, diagonal, increments[0], increments[1], increments[2], face);

	float cr_l = (total_length * cr_l_2_int) / ((float)diagonal);
	float li_l = (total_length * li_l_2_int) / ((float)diagonal);
	float lo_l = (total_length * lo_l_2_int) / ((float)diagonal);
	float bu_l = (total_length * bu_l_2_int) / ((float)diagonal);

	float absorption = 0;
	float li_absorption = 0;
	float lo_absorption = 0;
	float cr_absorption = 0;
	float bu_absorption = 0;
	s_absorption[threadIdx.x] = coeff_li * li_l + coeff_lo * lo_l + coeff_cr * cr_l + coeff_bu * bu_l;

	__syncthreads();
	absorption = Reduce_SM(s_absorption);

	Reduce_WARP(&absorption);

	__syncthreads();

	// calculation of the absorption for given ray
	if (threadIdx.x == 0)
	{
		// d_absorption[id] = absorption;
		d_result_list[index * len_coord_list * 2 + id] = absorption;
	}
}

__device__ void determine_boundaries(int *s_ray_classes, int offset, int *boundaries, int *class_values, int *boundary_count)
{
	int tid = threadIdx.x % warpSize; // Thread id within the warp

	int val = s_ray_classes[offset + tid];
	int prev_val;

	if (tid == 0)
	{
		prev_val = (threadIdx.x > 0) ? s_ray_classes[offset + tid - 1] : 3; // Fetch directly from shared memory
	}
	else
	{
		prev_val = __shfl_down_sync(0xFFFFFFFF, val, 1); // Get the value of the previous thread in the warp
	}

	if (threadIdx.x != 0 && val != prev_val)
	{

		int local_count = atomicAdd(boundary_count, 1);
		boundaries[local_count] = offset + tid + 1;
		class_values[local_count] = val;
	}
}

__device__ void calculate_distances(int *boundaries, int *class_values, int count, int *distances)
{
	for (int i = 0; i < count - 1; i++)
	{
		distances[i] = boundaries[i + 1] - boundaries[i];
	}
	distances[count - 1] = warpSize - boundaries[count - 1];
}

__global__ void rt_gpu_absorption_shuffle(int8_t *d_label_list, int *d_coord_list, int *d_face, float *d_angles, float *d_increments, float *d_result_list, int index)
{
	size_t id = blockIdx.x;
	int is_ray_incomming = id & 1;
	size_t pos = (id >> 1); /* the right shift operation effectively divided the value of id by 2 (since shifting the bits to the right by 1 is equivalent to integer division by 2).*/
	float increments[3];
	int face = 0;
	int coord[3];
	float theta, phi;
	// __shared__ float s_absorption[1024];
	// extern __shared__ int s_ray_classes[];
	// extern __shared__ int DynamicsharedMemory[];
	// int * s_ray_classes = DynamicsharedMemory;
	// int * boundaries = &DynamicsharedMemory[diagonal];
	// int * class_values = &DynamicsharedMemory[diagonal*2];

	// extern __shared__ int s_ray_classes[];

	__shared__ int s_ray_classes[2048];
	__shared__ int boundaries[128];
	__shared__ int class_values[128];

	__shared__ int boundary_count;
	// __shared__ float cr_l;
	// __shared__ float li_l;
	// __shared__ float bu_l;
	// __shared__ float lo_l;
	int cr_l = 0;
	int li_l = 0;
	int bu_l = 0;
	int lo_l = 0;
	int total_length;
	float absorption;

	if (threadIdx.x == 0)
	{
		boundary_count = 0;
		// cr_l = 0;
		// li_l = 0;
		// bu_l = 0;
		// lo_l = 0;
	}
	// extern __shared__ float s_absorption[];

	// Load coordinates
	coord[0] = d_coord_list[3 * pos + 0]; // z
	coord[1] = d_coord_list[3 * pos + 1]; // y
	coord[2] = d_coord_list[3 * pos + 2]; // x

	// Load face
	face = d_face[index * len_coord_list * 2 + id];
	// printf("index= %ld face=%d\n", index*len_coord_list*2 + id,face);

	// Load angle
	theta = d_angles[4 * blockIdx.y + 2 * is_ray_incomming];
	// phi = d_angles[4*blockIdx.y + 2*is_ray_incomming + 1];

	// Load Increment
	size_t incr_pos = 36 * blockIdx.y + 18 * is_ray_incomming + 3 * (face - 1);
	// get_increment_ratio(&increments[0], &increments[1], &increments[2], theta, phi, face);
	increments[0] = d_increments[incr_pos + 0];
	increments[1] = d_increments[incr_pos + 1];
	increments[2] = d_increments[incr_pos + 2];

	// Calculate number of iterations of blocks
	// trick for ceiling
	int nIter = (int)((diagonal + blockDim.x - 1) / blockDim.x);

	for (int f = 0; f < nIter; f++)
	{
		// calculate position based on threads id
		// check if the position is within a cube_face
		// write into ray_direction
		int lpos = (f * blockDim.x + threadIdx.x);
		int x, y, z;
		get_new_coordinates(
			&x, &y, &z,
			coord[2], coord[1], coord[0],
			increments[0], increments[1], increments[2],
			lpos, theta, face);
		int label = 0;

		if (
			x < x_max && y < y_max && z < z_max &&
			x >= 0 && y >= 0 && z >= 0)
		{
			size_t cube_pos = INDEX_3D(
				z_max, y_max, x_max,
				z, y, x);
			label = (int)d_label_list[cube_pos];

			if (lpos < diagonal)
			{
				s_ray_classes[lpos] = label;
			}
		}
	}

	// __shared__ int distances[32 * warpSize];
	// int count = 0;

	__syncthreads();
	for (int f = 0; f < nIter; f++)
	{
		int lpos = (f * blockDim.x + threadIdx.x);
		int warpId = lpos / warpSize;
		int laneId = lpos % warpSize;
		if (lpos < diagonal)
		{
			determine_boundaries(s_ray_classes, warpId * warpSize, boundaries, class_values, &boundary_count);
		}
	}
	__syncthreads();
	if (blockIdx.x == 100)
	{
		printf("boundary_count=%d\n", boundary_count);
	}
	if (threadIdx.x == 0)
	{
		for (int count; count < boundary_count; count++)
		{
			if (count == 0)
			{
				cr_l += boundaries[count];
			}
			else
			{
				int distance = boundaries[count] - boundaries[count - 1];
				if (class_values[count] == 3)
					cr_l += distance;
				else if (class_values[count] == 1)
					li_l += distance;
				else if (class_values[count] == 2)
					lo_l += distance;
				else if (class_values[count] == 4)
					bu_l += distance;
				else
				{
				}
			}
		}
		absorption = coeff_li * li_l + coeff_lo * lo_l + coeff_cr * cr_l + coeff_bu * bu_l;
		d_result_list[index * len_coord_list * 2 + id] = absorption;
	}
	// Only a fraction of the threads will calculate distances to reduce redundant work.
	// if (laneId < boundary_count) {
	//     calculate_distances(boundaries + warpId * 32, class_values + warpId * 32, boundary_count, distances + warpId * 32);
	// }
	// get_distance_2(&total_length, diagonal, increments[0], increments[1], increments[2], face);

	// float cr_l = (total_length * cr_l_2_int) / ((float)diagonal);
	// float li_l = (total_length * li_l_2_int) / ((float)diagonal);
	// float lo_l = (total_length * lo_l_2_int) / ((float)diagonal);
	// float bu_l = (total_length * bu_l_2_int) / ((float)diagonal);

	// float absorption = 0;
	// float li_absorption = 0;
	// float lo_absorption = 0;
	// float cr_absorption = 0;
	// float bu_absorption = 0;
	// s_absorption[threadIdx.x] = coeff_li * li_l + coeff_lo * lo_l + coeff_cr * cr_l + coeff_bu * bu_l;

	// __syncthreads();
	// absorption = Reduce_SM(s_absorption);

	// Reduce_WARP(&absorption);

	// __syncthreads();

	// calculation of the absorption for given ray

	// }
}

float *ray_tracing_gpu_single(int rotated_s1_size, int rotated_xray_size, int h_len_result,int h_x_max, int  h_y_max, int h_z_max, int h_diagonal, int h_len_coord_list, float *coefficients, float * voxel_size, size_t result_size, size_t python_result_size,size_t scattering_vector_list_size,size_t omega_list_size,size_t raw_xray_size,size_t omega_axis_size, size_t kp_rotation_matrix_size, size_t coord_list_size, size_t cube_size,size_t face_size,size_t angle_size,size_t increments_size,size_t ray_classes_size,size_t absorption_size,float* h_result_list, const float *scattering_vector_list,const float *omega_list,const float *raw_xray,const float *omega_axis,const float *kp_rotation_matrix,int *coord_list,int8_t *label_list_1d,float * h_python_result_list){
			//----------> Memory allocation
		//---------> Allocating memory on the device
		// global memory
		hipError_t hipError_t;
		int nCUDAErrors = 0;
		float *d_result_list;		 // contains i reflections, each with j rays
		float *d_python_result_list; // contains i reflections
		float *d_scattering_vector_list;
		float *d_omega_list;
		float *d_raw_xray;
		float *d_omega_axis;
		float *d_kp_rotation_matrix;
		float *d_rotated_s1_list;
		float *d_rotated_xray_list;
		int *d_coord_list;
		int8_t *d_label_list;

		// individual memory for each relfection
		int *d_face;
		float *d_angles;
		float *d_increments;
		int *d_ray_classes;
		float *d_absorption_lengths;

		// output memory
		float *h_rotated_s1_list = (float *)malloc(rotated_s1_size);
		float *h_rotated_xray_list = (float *)malloc(rotated_xray_size);
		// float *h_result_list = (float *)malloc(result_size);

		/* creating  global memory for constants */
		// int   x_max,y_max,z_max,diagonal,len_coord_list, len_result;
		// float coeff_cr, coeff_bu, coeff_lo, coeff_li,voxel_length_x,voxel_length_y,voxel_length_z;

		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(len_result), &h_len_result, sizeof(h_len_result)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(x_max), &h_x_max, sizeof(h_x_max)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(y_max), &h_y_max, sizeof(h_y_max)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(z_max), &h_z_max, sizeof(h_z_max)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(diagonal), &h_diagonal, sizeof(h_diagonal)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(len_coord_list), &h_len_coord_list, sizeof(h_len_coord_list)));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(coeff_li), &coefficients[0], sizeof(coefficients[0])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(coeff_lo), &coefficients[1], sizeof(coefficients[1])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(coeff_cr), &coefficients[2], sizeof(coefficients[2])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(coeff_bu), &coefficients[3], sizeof(coefficients[3])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxel_length_z), &voxel_size[0], sizeof(voxel_size[0])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxel_length_y), &voxel_size[1], sizeof(voxel_size[1])));
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(voxel_length_x), &voxel_size[2], sizeof(voxel_size[2])));
		printf("result_size: %ld\n", result_size);
		hipError_t = hipMalloc((void **)&d_result_list, result_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_result_list\n");
			d_result_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_python_result_list, python_result_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_python_result_list\n");
			d_result_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_scattering_vector_list, scattering_vector_list_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_scattering_vector_list\n");
			d_scattering_vector_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_omega_list, omega_list_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_omega_list\n");
			d_omega_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_raw_xray, raw_xray_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_raw_xray\n");
			d_raw_xray = NULL;
		}
		hipError_t = hipMalloc((void **)&d_omega_axis, omega_axis_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_omega_axis\n");
			d_omega_axis = NULL;
		}
		hipError_t = hipMalloc((void **)&d_kp_rotation_matrix, kp_rotation_matrix_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_kp_rotation_matrix\n");
			d_kp_rotation_matrix = NULL;
		}

		hipError_t = hipMalloc((void **)&d_rotated_xray_list, rotated_xray_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_rotated_xray_list\n");
			d_coord_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_rotated_s1_list, rotated_s1_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_rotated_s1_list\n");
			d_coord_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_coord_list, coord_list_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_coord_list\n");
			d_coord_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_label_list, cube_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_label_list\n");
			d_label_list = NULL;
		}

		hipError_t = hipMalloc((void **)&d_face, face_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_face\n");
			d_label_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_angles, angle_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_angles\n");
			d_label_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_increments, increments_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_increments\n");
			d_label_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_ray_classes, ray_classes_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_ray_classes\n");
			d_label_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_absorption_lengths, absorption_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_absorption_lengths \n");
			d_label_list = NULL;
		}
		hipError_t = hipMalloc((void **)&d_absorption_lengths, absorption_size);
		if (hipError_t != hipSuccess)
		{
			nCUDAErrors++;
			printf("ERROR: memory allocation d_absorption_lengths \n");
			d_label_list = NULL;
		}
		//---------> Memory copy and preparation
		GpuTimer timer;
		float memory_time = 0;
		timer.Start();

		hipError_t = hipMemcpy(d_scattering_vector_list, scattering_vector_list, scattering_vector_list_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_scattering_vector_list.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_omega_list, omega_list, omega_list_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_omega_list.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_raw_xray, raw_xray, raw_xray_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_raw_xray.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_omega_axis, omega_axis, omega_axis_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_omega_axis.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_kp_rotation_matrix, kp_rotation_matrix, kp_rotation_matrix_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_kp_rotation_matrix.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_coord_list, coord_list, coord_list_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_coord_list.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		hipError_t = hipMemcpy(d_label_list, label_list_1d, cube_size, hipMemcpyHostToDevice);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Memcopy d_label_list.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		// hipError_t = hipMemset(d_ray_classes, 0, ray_classes_size);
		// if (hipError_t != hipSuccess)
		// {
		// 	printf("ERROR! Memset d_ray_classes.\n");
		// 	print_cuda_error(hipError_t);
		// 	nCUDAErrors++;
		// }
		timer.Stop();
		memory_time = timer.Elapsed();
		float single_time = memory_time;
		printf("Total time: %0.3f ms; memory_time : %0.3f ms\n", single_time, memory_time);
		hipDeviceSynchronize();

		//---------> Kernel execution
		float precompute_time = 0;
		printf("len result: %d\n", h_len_result);
		timer.Start();
		if (nCUDAErrors == 0)
		{

			{
				int nThreads = 256;
				int nBlocks = (h_len_result + nThreads - 1) / nThreads;
				dim3 gridSize_rotation(nBlocks, 1, 1);
				dim3 blockSize_rotation(nThreads, 1, 1);

				ray_tracing_rotation<<<gridSize_rotation, blockSize_rotation>>>(d_omega_axis, d_omega_list, d_kp_rotation_matrix, d_raw_xray, d_scattering_vector_list, d_rotated_xray_list, d_rotated_s1_list);
			}

			// {
			// 	int nThreads_x = 256;
			// 	int nThreads_y = 4;
			// 	int nBlocks_x = ((h_len_coord_list * 2) + nThreads_x - 1) / nThreads_x;
			// 	int nBlocks_y = (h_len_result + nThreads_y - 1) / nThreads_y;
			// 	dim3 gridSize_face(nBlocks_x, nBlocks_y);
			// 	dim3 blockSize_face(nThreads_x, nThreads_y);
			// 	printf("nBlocks_x: %d\n", nBlocks_x);
			// 	printf("nBlocks_y: %d\n", nBlocks_y);
			// 	printf("nThreads_x: %d\n", nThreads_x);
			// 	printf("nThreads_y: %d\n", nThreads_y);
			// 	rt_gpu_get_face_overall<<<gridSize_face, blockSize_face>>>(
			// 		d_face,
			// 		d_coord_list,
			// 		d_rotated_s1_list,
			// 		d_rotated_xray_list);
			// 	// output_size = h_len_coord_list * 2 * h_len_result; 0101=>s1,s0,s1,s0
			// }
		}
		//---------> error check
		hipError_t = hipGetLastError();
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! GPU Kernel error.\n");
			print_cuda_error(hipError_t);
			nCUDAErrors++;
		}
		else
		{
			if (DEBUG)
			{
				printf("No CUDA error.\n");
				printf("Rotation matrices are calculated\n");
			}
		}
		//-----> Copy chunk of output data to host
		// hipError_t = hipMemcpy(h_rotated_xray_list, d_rotated_xray_list, rotated_xray_size, hipMemcpyDeviceToHost);
		// if (hipError_t != hipSuccess)
		// {
		// 	printf("ERROR! Copy of d_rotated_xray_list has failed.\n");
		// 	nCUDAErrors++;
		// }
		// hipError_t = hipMemcpy(h_rotated_s1_list, d_rotated_s1_list, rotated_s1_size, hipMemcpyDeviceToHost);
		// if (hipError_t != hipSuccess)
		// {
		// 	printf("ERROR! Copy of d_rotated_s1_list has failed.\n");
		// 	nCUDAErrors++;
		// }
		hipDeviceSynchronize();
		timer.Stop();
		precompute_time = timer.Elapsed();
		single_time += precompute_time;
		printf("Total time: %0.3f ms; Precompute time: %0.3f ms\n", single_time, precompute_time);

		float compute_time = 0;
		timer.Start();

		for (int i = 0; i < (int)h_len_result; i++)
		{

			if (nCUDAErrors == 0)

			{

				//**************** Calculate faces **************
				{
					int nThreads = 128;
					int nBlocks = ((h_len_coord_list * 2) + nThreads - 1) / nThreads;

					dim3 gridSize_face(nBlocks, 1, 1);
					dim3 blockSize_face(nThreads, 1, 1);
					rt_gpu_get_face<<<gridSize_face, blockSize_face>>>(
						d_face,
						d_coord_list,
						d_rotated_s1_list,
						d_rotated_xray_list, i);
				}

				{
					int nBatches = 1;
					int nThreads = 128;
					int nBlocks = ((nBatches * 2) + nThreads - 1) / nThreads;
					dim3 gridSize_face(nBlocks, 1, 1);
					dim3 blockSize_face(nThreads, 1, 1);
					rt_gpu_angles<<<gridSize_face, blockSize_face>>>(
						d_angles,
						d_rotated_s1_list,
						d_rotated_xray_list,
						nBatches, i);
				}

				{
					int nBatches = 1;
					int nThreads = 12;
					dim3 gridSize_face(nBatches, 1, 1);
					dim3 blockSize_face(nThreads, 1, 1);
					rt_gpu_increments<<<gridSize_face, blockSize_face>>>(
						d_increments,
						d_angles);
				}

				//---------> error check
				hipError_t = hipGetLastError();
				if (hipError_t != hipSuccess)
				{
					printf("ERROR! GPU Kernel error.\n");
					print_cuda_error(hipError_t);
					nCUDAErrors++;
				}
				// else
				// {
				// 	printf("No CUDA error.\n");
				// }

				{
					// float voxel_length_z = voxel_size[0];
					// float voxel_length_y = voxel_size[1];
					// float voxel_length_x = voxel_size[2];
					// float coeff_li = coefficients[0];
					// float coeff_lo = coefficients[1];
					// float coeff_cr = coefficients[2];
					// float coeff_bu = coefficients[3];

					int nBlocks = h_len_coord_list * 2; // one block for one crystal voxel
					int nThreads = 256;					// 256:49s ,128:49s 32:52s,512:fail,320:55s
					dim3 gridSize_face(nBlocks, 1, 1);
					dim3 blockSize_face(nThreads, 1, 1);

					rt_gpu_absorption<<<gridSize_face, blockSize_face>>>(

						d_label_list,
						d_coord_list,
						d_face,
						d_angles,
						d_increments, d_result_list, i); //,sizeof(int)*h_diagonal*3

					// rt_gpu_absorption_shuffle<<<gridSize_face, blockSize_face>>>(
					// 	d_label_list,
					// 	d_coord_list,
					// 	d_face,
					// 	d_angles,
					// 	d_increments, d_result_list, i);
				}
			}

			hipDeviceSynchronize();

			if (i % 1000 == 0)
			{
				timer.Stop();
				compute_time = timer.Elapsed();
				single_time += compute_time;
				printf("--> Batch [%d]: total time: %0.3fms; Compute time: %0.3fms;\n", i, single_time, compute_time);
				timer.Start();
			}
		}
		//---------> summing the results and output the final array

		int nThreads = 256;
		int nBlocks = (h_len_result + nThreads - 1) / nThreads;
		dim3 gridSize_face(nBlocks, 1, 1);
		dim3 blockSize_face(nThreads, 1, 1);
		rt_gpu_python_results<<<gridSize_face, blockSize_face>>>(d_result_list, d_python_result_list, h_len_result);

		hipError_t = hipMemcpy(h_python_result_list, d_python_result_list, python_result_size, hipMemcpyDeviceToHost);
		if (hipError_t != hipSuccess)
		{
			printf("ERROR! Copy of d_python_result_list has failed.\n");
			nCUDAErrors++;
		}
		// printf("Copying from device to host:\n");
		// printf("  Size: %zu\n", python_result_size);
		// printf("  Device pointer: %p\n", d_python_result_list);
		// printf("  Host pointer: %p\n", h_python_result_list);

		printf("Total time spent is: %fms\n", single_time);
		//-----> Free memory
		hipFree(d_result_list);
		hipFree(d_python_result_list);
		hipFree(d_scattering_vector_list);
		hipFree(d_omega_list);
		hipFree(d_raw_xray);
		hipFree(d_omega_axis);
		hipFree(d_kp_rotation_matrix);
		hipFree(d_rotated_s1_list);
		hipFree(d_rotated_xray_list);
		hipFree(d_coord_list);
		hipFree(d_label_list);
		hipFree(d_face);
		hipFree(d_angles);
		hipFree(d_increments);
		hipFree(d_ray_classes);
		hipFree(d_absorption_lengths);
		free(h_rotated_s1_list);
		free(h_rotated_xray_list);

		return h_python_result_list;
}


void transpose(float *input, int rows, int cols, float *output);
void dot_product(const float *A, const float *B, float *C, int m, int n, int p);
void kp_rotation(const float *axis, float theta, float *result);

// size_t multiplier(int multiplier_1, int multiplier_2)
// {
// 	return multiplier_1 * multiplier_2;
// }

int ray_tracing_gpu_overall_kernel(int low, int up,
								   int *coord_list,
								   int h_len_coord_list,
								   const float *scattering_vector_list, const float *omega_list,
								   const float *raw_xray,
								   const float *omega_axis, const float *kp_rotation_matrix,
								   int h_len_result,
								   float *voxel_size, float *coefficients,
								   int8_t *label_list_1d, int *shape, int full_iteration,
								   int store_paths, float *h_result_list, int *h_face, float *h_angles, float *h_python_overall_result_list)
{
	//---------> Initial nVidia stuff
	int devCount;
	hipError_t hipError_t;
	hipError_t = hipGetDeviceCount(&devCount);
	if (hipError_t != hipSuccess || devCount == 0)
	{
		printf("ERROR: CUDA capable device not found!\n");
		return (1);
	}

	printf("--> GPU info:");

	int deviceID = 0; // Replace with the desired device ID
	hipSetDevice(deviceID);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, deviceID);

	unsigned long sharedMemPerBlock = deviceProp.sharedMemPerBlock;
	if (deviceProp.major >= 1)
	{
		printf("Device %d: %s\n", deviceID, deviceProp.name);
		printf("Shared memory per block: %lu bytes\n", sharedMemPerBlock);
	}
	
	//---------> Checking memory
	size_t free_mem, total_mem;
	hipMemGetInfo(&free_mem, &total_mem);
	printf("--> GPU info: Device has %0.3f MB of total memory, which %0.3f MB is available.\n", ((float)total_mem) / (1024.0 * 1024.0), (float)free_mem / (1024.0 * 1024.0));
	float total_time = 0;
	int n_chunks = 1;
	int h_z_max = shape[0], h_y_max = shape[1], h_x_max = shape[2];
	int64_t h_len_result_coord=(int64_t)h_len_result * (int64_t)h_len_coord_list;

	int h_diagonal = h_x_max * sqrtf(3);
	size_t cube_size = h_x_max * h_y_max * h_z_max * sizeof(int8_t);
	size_t face_size =  h_len_coord_list * 2 * sizeof(int);
	size_t absorption_size = h_len_coord_list * 2 * sizeof(float);
	size_t angle_size = 4 * sizeof(float);
	size_t increments_size = 36 * sizeof(float);
	size_t ray_classes_size = h_diagonal * h_len_coord_list * 2 * sizeof(int);
	size_t coord_list_size = h_len_coord_list * 3 * sizeof(int);
	size_t ray_directions_size = 3 * sizeof(float);

	// size_t   result_size = multiplier(h_len_result,h_len_coord_list) * h_len_coord_list * 2 * sizeof(float);
	size_t result_size = h_len_result_coord * 2 * sizeof(float);
	size_t python_result_size = h_len_result * sizeof(float); // my desktop doesnt have enough memory to store the whole result list, so take a half of it to test
	size_t scattering_vector_list_size = h_len_result * sizeof(float) * 3;
	size_t omega_list_size = h_len_result * sizeof(float);
	size_t raw_xray_size = 3 * sizeof(float);
	size_t omega_axis_size = 3 * sizeof(float);
	size_t kp_rotation_matrix_size = 9 * sizeof(float);
	size_t rotated_s1_size = h_len_result * sizeof(float) * 3;
	size_t rotated_xray_size = h_len_result * sizeof(float) * 3;
	// size_t face_size = h_len_result * h_len_coord_list * 2 * sizeof(int);
	printf("len_coord_list %d \n", h_len_coord_list);
	printf("h_len_result %d \n", h_len_result);
	size_t  total_memory_required_bytes = face_size + angle_size + increments_size + absorption_size + cube_size + ray_classes_size + coord_list_size + ray_directions_size + result_size + scattering_vector_list_size + omega_list_size + raw_xray_size + omega_axis_size + kp_rotation_matrix_size + rotated_s1_size + rotated_xray_size;

	printf("total_memory_required_bytes %f \n", total_memory_required_bytes);
	printf("--> DEBUG: Total memory required %0.3f MB.\n", (double)total_memory_required_bytes / (1024.0 * 1024.0));

	if (total_memory_required_bytes > free_mem)
	{
		printf("--> DEBUG: Total memory required %0.3f MB.\n", (double)
																	   total_memory_required_bytes /
																   (1024.0 * 1024.0));
		printf(" Not enough memory! Input data is splitted into chunks.\n");
		// return (1);
		n_chunks = (total_memory_required_bytes + free_mem - 1) / free_mem;
	}
	else
	{
	}
	
	float *h_python_result_list = (float *)malloc(python_result_size);
	for (int chunk = 0; chunk < n_chunks; chunk++)
	{
		h_python_overall_result_list=ray_tracing_gpu_single(rotated_s1_size, rotated_xray_size, h_len_result, h_x_max, h_y_max, h_z_max, h_diagonal, h_len_coord_list, coefficients, voxel_size, result_size, python_result_size, scattering_vector_list_size, omega_list_size, raw_xray_size, omega_axis_size, kp_rotation_matrix_size, coord_list_size, cube_size, face_size, angle_size, increments_size, ray_classes_size, absorption_size, h_result_list, scattering_vector_list, omega_list, raw_xray, omega_axis, kp_rotation_matrix, coord_list, label_list_1d, h_python_result_list);

	}

	return (0);
}